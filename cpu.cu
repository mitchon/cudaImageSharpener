#include <stdio.h>
#include "Targa.h"
#include <hip/hip_runtime.h>
#include <>

int main(int argc, char **argv)
{
    if (argc!=3)
        return 1;
    unsigned char *data;
    unsigned w=0, h=0, bpp=0;
    if(Targa2Array(argv[1],&data,&w,&h,&bpp) == 1)
        return 1;
    
    if (bpp/8 == 3)
        TrueColorToMonochrome(&data, 0, w, h, &bpp);

    float a = 0.1;
	printf("Введите a: ");
	scanf("%f", &a);

    float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    
    int x = w/5+((w%5)!=0);
    int y = h/5+((h%5)!=0);

    hipEventRecord(start, 0);
    
    int yIndex, xIndex;

    unsigned char *unsharp=(unsigned char *)malloc(h * w);
    for (int i = 0; i < y; i++) {
        for (int j = 0; j < x; j++) {
            int avg = 0;
            for (int yCounter = 0; yCounter < 5; yCounter++) {
                if (yCounter+i*5 < h) yIndex = yCounter + i*5;
                else yIndex = h - 1;
                int horSum = 0;
                for (int xCounter = 0; xCounter < 5; xCounter++) {
                    if (xCounter+j*5 < w) xIndex = xCounter + j*5;
                    else xIndex = w - 1;
                    horSum += data[yIndex*w+xIndex];
                }
                avg += horSum;
            }
            avg = avg / 25;

            for (int yCounter = 0; yCounter < 5; yCounter++)
                if (yCounter+i*5 < h)
                    for (int xCounter = 0; xCounter < 5; xCounter++)
                        if (xCounter+j*5 < w) {
                            int substraction = (int)data[(yCounter+i*5)*w+(xCounter+j*5)] - avg;
                            if (substraction < 0) substraction = 0;
                            int sum = (int)data[(yCounter+i*5)*w+(xCounter+j*5)]+(int)(a * substraction);
                            if (sum > 255) sum = 255;
                            unsharp[(yCounter+i*5)*w+(xCounter+j*5)]=sum;                 
                        }
        }
    }
    
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
    
    Array2Targa(argv[2],unsharp,w,h,bpp);
    free(data);
    free(unsharp);

    printf("Elapsed time (ms) = %f\n", elapsedTime);
    return 0;
}
