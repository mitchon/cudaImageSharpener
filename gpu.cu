#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "Targa.h"

__shared__ unsigned char data[10000], unsharp[10000];


__global__ void FUN_KERNEL(unsigned char* datag, unsigned char* unsharpg, int w, int h, int x, int y, float a, int S, int T)
{
	int idx_thread = blockIdx.x * blockDim.x + threadIdx.x;//номер потока в задаче
    int yIndex, xIndex;
    int tmp = T;
    if (tmp > x)
        tmp = x;

    for (int sy = 0; sy < y; sy++) {
        int i = sy;
        for (int sx = 0; sx < S; sx++) {
            int jg = sx * T + idx_thread;
            int j = idx_thread;
            if (jg < x) {
                for (int yCounter = 0; yCounter < 5; yCounter++)
                    if (yCounter+i*5 < h)
                        for (int xCounter = 0; xCounter < 5; xCounter++)
                            if (xCounter+jg*5 < w)
                                data[yCounter*tmp*5+(xCounter+j*5)] = datag[(yCounter+i*5)*w+(xCounter+jg*5)];

                int avg = 0;
                for (int yCounter = 0; yCounter < 5; yCounter++) {
                    if (yCounter+i*5 < h) yIndex = yCounter;
                    else yIndex = (h - 1) % 5;
                    int horSum = 0;
                    for (int xCounter = 0; xCounter < 5; xCounter++) {
                        if (xCounter+jg*5 < w) xIndex = xCounter + j*5;
                        else xIndex = (w - 1) % 5;
                        horSum += data[yIndex*tmp*5+(xIndex)];
                    }
                    avg += horSum;
                }
                avg = avg / 25;

                for (int yCounter = 0; yCounter < 5; yCounter++)
                    if (yCounter+i*5 < h)
                        for (int xCounter = 0; xCounter < 5; xCounter++)
                            if (xCounter+jg*5 < w) {
                                int substraction = (int)data[yCounter*tmp*5+(xCounter+j*5)] - avg;
                                if (substraction < 0) substraction = 0;
                                int sum = (int)data[yCounter*tmp*5+(xCounter+j*5)]+(int)(a * substraction);
                                if (sum > 255) sum = 255;
                                unsharp[yCounter*tmp*5+(xCounter+j*5)]=sum; 
                            }

                for (int yCounter = 0; yCounter < 5; yCounter++)
                    if (yCounter+i*5 < h)
                        for (int xCounter = 0; xCounter < 5; xCounter++)
                            if (xCounter+jg*5 < w)
                                unsharpg[(yCounter+i*5)*w+(xCounter+jg*5)] = unsharp[yCounter*tmp*5+(xCounter+j*5)];
            }
        }

    }
}

int main(int argc, char **argv)
{

    if (argc!=3)
        return 1;
    int blocks, blocksize, steps, threadsTotal;
    unsigned char *devData;
    unsigned char *data;
    unsigned char *devUnsharp;
    unsigned char *unsharp;

    unsigned w=0, h=0, bpp=0;
    if(Targa2Array(argv[1],&data,&w,&h,&bpp) == 1)
        return 1;

    if (bpp/8 == 3)
        TrueColorToMonochrome(&data, 0, w, h, &bpp);

    float a = 0.1;
	printf("Введите a: ");
	scanf("%f", &a);
    
    float elapsedTime, copyTime1, copyTime2, deviceTime;
	hipEvent_t start, stop, event1, event2;
	hipEventCreate(&start);
	hipEventCreate(&event1);
	hipEventCreate(&event2);
	hipEventCreate(&stop);

	printf("Введите количество блоков: ");
	scanf("%i", &blocks);
	printf("Введите количество нитей: ");
	scanf("%i", &blocksize);

    unsigned int ImageSize = w*h;

    int x = w/5+((w%5)!=0);
    int y = h/5+((h%5)!=0);
	threadsTotal = blocks * blocksize;
    steps = x / threadsTotal + ((x % threadsTotal) != 0);
    printf("X: %d\n", x);
    printf("S: %d\n", steps);

	hipMalloc((void**)&devData, ImageSize * sizeof(unsigned char));
	hipMalloc((void**)&devUnsharp, ImageSize * sizeof(unsigned char));
    unsharp = (unsigned char *)malloc(ImageSize);

    hipEventRecord(start, 0);
    hipMemcpy(devData, data, ImageSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipEventRecord(event1, 0);

    FUN_KERNEL <<< blocks, blocksize >>> (devData, devUnsharp, w, h, x, y, a, steps, threadsTotal);
    
    hipEventRecord(event2, 0);
    hipMemcpy(unsharp, devUnsharp, ImageSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventElapsedTime(&copyTime1, start, event1);
	hipEventElapsedTime(&deviceTime, event1, event2);
	hipEventElapsedTime(&copyTime2, event2, stop);
    
    Array2Targa(argv[2],unsharp,w,h,bpp);

	hipFree(devData);
	hipFree(devUnsharp);
    
    free(data);
    free(unsharp);

    printf("Elapsed time (ms) =\t%f\n", elapsedTime);
    printf("Copy time (ms) =\t%f\n", copyTime1 + copyTime2);
    printf("Device time (ms) =\t%f\n", deviceTime);
    return 0;
}